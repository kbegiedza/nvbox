#ifndef NVBOX_UTILITY_HPP_
#define NVBOX_UTILITY_HPP_

#include <cstdio>
#include <string>
#include <nvbox/utility.hpp>
#include <hip/hip_runtime.h>

void nvbox::describeCuda()
{
    printf("Quering CUDA devices...\n");

    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess)
    {
        printf("Cannot get device count.\n[ERR %d] => %s\n",
               static_cast<int>(error),
               hipGetErrorString(error));

        return;
    }

    printf("CUDA devices:\t\t%d\n", deviceCount);

    int driverVersion, runtimeVersion;

    hipDriverGetVersion(&driverVersion);
    hipDriverGetVersion(&runtimeVersion);

    auto getCudaVersionString = [&](int version) -> std::string
    {
        return std::to_string(version / 1000) + "." + std::to_string((version % 100) / 10);
    };

    printf("CUDA Driver Version:\t%s\nRuntime Version:\t%s\n",
           getCudaVersionString(driverVersion).c_str(),
           getCudaVersionString(runtimeVersion).c_str());
}
#endif