#include <cstdio>
#include <string>

#include "nvbox/utility.cuh"

void nvbox::describeCuda()
{
    int driverVersion, runtimeVersion;

    hipDriverGetVersion(&driverVersion);
    hipDriverGetVersion(&runtimeVersion);

    auto getCudaVersionString = [&](int version) -> std::string
    {
        return std::to_string(version / 1000) + "." + std::to_string((version % 100) / 10);
    };

    printf("CUDA Driver Version:\t%s\nRuntime Version:\t%s\n",
           getCudaVersionString(driverVersion).c_str(),
           getCudaVersionString(runtimeVersion).c_str());
}

void nvbox::describeCudaDevices()
{
    int deviceCount = 0;
    handleCudaError(hipGetDeviceCount(&deviceCount));

    printf("CUDA devices:\t\t%d\n", deviceCount);

    for (int deviceId = 0; deviceId < deviceCount; ++deviceId)
    {
        printf("===\nDevice:\t%d\n", deviceId);

        int attributeValue;

        handleCudaError(hipDeviceGetAttribute(&attributeValue, hipDeviceAttribute_t::hipDeviceAttributeMaxThreadsPerBlock, deviceId));
        printf("MaxThreadsPerBlock:\t%d\n", attributeValue);

        handleCudaError(hipDeviceGetAttribute(&attributeValue, hipDeviceAttribute_t::hipDeviceAttributeMaxThreadsPerMultiProcessor, deviceId));
        printf("MaxThreadsPerMultiProcessor:\t%d\n", attributeValue);

        handleCudaError(hipDeviceGetAttribute(&attributeValue, hipDeviceAttribute_t::hipDeviceAttributeClockRate, deviceId));
        printf("ClockRate:\t%d\n", attributeValue);

        handleCudaError(hipDeviceGetAttribute(&attributeValue, hipDeviceAttribute_t::hipDeviceAttributeMemoryClockRate, deviceId));
        printf("MemoryClockRate:\t%d\n", attributeValue);

        hipDeviceProp_t deviceProp;
        handleCudaError(hipGetDeviceProperties(&deviceProp, deviceId));
    }
}

void nvbox::handleCudaError(hipError_t error)
{
    if (error != hipSuccess)
    {
        int errorCode = static_cast<int>(error);
        const char *errorString = hipGetErrorString(error);

        printf("Cannot get device count.\n[ERR %d] => %s\n", errorCode, errorString);

        exit(EXIT_FAILURE);
    }
}