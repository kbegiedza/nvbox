#include "hip/hip_runtime.h"
#include "nvbox/demo.cuh"

#include "hip/hip_runtime.h"
#include ""

namespace nvbox
{
    __global__ void VecAdd(float *a, float *b, float *c, int n)
    {
        int i = blockDim.x * blockDim.x + threadIdx.x;

        if (i < n)
        {
            *(c + i) = *(a + i) + *(b + i);
        }
    }

    void RunAddDemo()
    {
        int N = 100;
        size_t size = N * sizeof(float);

        float *hostA = (float *)malloc(size);
        float *hostB = (float *)malloc(size);
        float *hostC = (float *)malloc(size);

        float *deviceA;
        float *deviceB;
        float *deviceC;

        hipMalloc(&deviceA, size);
        hipMalloc(&deviceB, size);
        hipMalloc(&deviceC, size);

        hipMemcpy(deviceA, hostA, size, hipMemcpyHostToDevice);
        hipMemcpy(deviceB, hostB, size, hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        nvbox::VecAdd<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, N);

        hipMemcpy(hostC, deviceC, size, hipMemcpyDeviceToHost);

        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);

        free(hostA);
        free(hostB);
        free(hostC);
    }
}